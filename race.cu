#include <hip/hip_runtime.h>
#include <iostream>
#include <time.h>
#include <string>
#include <vector>
#include <algorithm>
#include <unistd.h>

using namespace std;

class Runner {       
  public:             
    double speed;
    double currentPosition;
    string name;
    bool isFinished;
    double finishTime;

    __host__ Runner();
    __host__ ~Runner();
    __host__ Runner(string nameOfRunner);
};

__host__ Runner::Runner()
{
        
}
__host__ Runner::~Runner()
{
        
}
__host__ Runner::Runner(string nameOfRunner)
{
    speed = 4.0 * ( (double)rand() / (double)RAND_MAX ) + 1.0;
    currentPosition = 0.0;
    name = "Kosucu " + nameOfRunner;
    isFinished = false;
    finishTime = 0;
}

class Race {       
  public:             
    int runwayLength;        
    int numberOfRunners;
    int timePassed;
    bool isFirstFinished;
    bool isRaceFinished;
    Runner* runners;

    __host__ Race(int lenght, int runnerNumber);
    __host__ Race();
    __host__ ~Race();
    __device__ void calculateNewPosition(int idx);
    __host__ void check();
    __host__ void printAllRunners(bool position);
    __host__ void sortRunners();
};

__host__ Race::Race(int lenght, int runnerNumber){
    runwayLength = lenght;
    numberOfRunners = runnerNumber;
        
    hipMallocManaged(&runners, runnerNumber*sizeof(Runner));
        
    for (int i = 0; i < numberOfRunners; i++) {
        runners[i] = Runner(to_string(i+1));
    }
        
    isFirstFinished = false;
    isRaceFinished = false;
    timePassed = 0;
         
}

__host__ Race::Race()
{
}

__host__ Race::~Race()
{
}
    
__device__ void Race::calculateNewPosition(int idx)
{
	if(!runners[idx].isFinished){
        runners[idx].currentPosition += runners[idx].speed;
    }
}
    
__host__ void Race::check()
{
    bool isFirstFinishedTemp = isFirstFinished;
    isRaceFinished = true;
    for (int i = 0; i < numberOfRunners; i++) {
        if(runners[i].currentPosition > runwayLength && !runners[i].isFinished){
            runners[i].isFinished = true;
            runners[i].finishTime = timePassed - (double)(runners[i].currentPosition-runwayLength)/runners[i].speed;
            runners[i].currentPosition = runwayLength;
                
            if(!isFirstFinished){
                isFirstFinished = true;
            }
        }
        if(!runners[i].isFinished)
            isRaceFinished = false;
    }
    if(!isFirstFinishedTemp && isFirstFinished){
        cout<<"***Bitis cizgisine ilk kosucu ulasti!***\n";
        cout<<"-Kosucularin anlik konumlari:\n";
        printAllRunners(true);
    }
        
}
    
__host__ void Race::printAllRunners(bool position)
{
    for(int i=0;i<numberOfRunners; i++){
        if(position)
            cout<< runners[i].name <<"; Konumu: "<<runners[i].currentPosition<<"m \n";
        else
            cout<<i+1<<". Yarisci: " <<runners[i].name <<"; Hizi: " <<runners[i].speed<<" m/s, Bitirme suresi: "<<runners[i].finishTime<<"s\n";
    }
}
    
__host__ void Race::sortRunners()
{
    for(int i=0;i<numberOfRunners; i++){
        for(int j=0;j<numberOfRunners-1; j++){
            if(runners[j].finishTime > runners[j+1].finishTime){
                Runner temp = runners[j];
                runners[j] = runners[j+1];
                runners[j+1] = temp;
            }
                
        }
    }
}


__global__ void raceKernel(Race race)
{
	const int idx = blockIdx.x*blockDim.x + threadIdx.x;
	if(idx < race.numberOfRunners)
	{
		race.calculateNewPosition(idx);
	}

}


int main()
{
    srand(time(NULL)); 
    int runwayLength = 100;
    int runnerNumber = 100;
    Race race = Race(runwayLength,runnerNumber);
    int currentTime = 0;
    cout<<"****Yaris basladi!****\n\n";
    while(!race.isRaceFinished)
    {
	cout<<"Gecen sure: "<<currentTime++<<"s\n";
    	raceKernel<<<1,runnerNumber>>>(race);
    	hipDeviceSynchronize();
        
        race.timePassed = race.timePassed + 1;
        race.check();

        usleep(1000000);
    }
    
    cout<<"\n****Yaris bitti!****\n";
    cout<<"-Siralama:\n";
    
    race.sortRunners();
    race.printAllRunners(false);
    
    hipFree(race.runners);
    
    return 0;
}
